#include "hip/hip_runtime.h"
#pragma region LICENSE

// Read the Gnu General Public License v3.0 at: https://www.gnu.org/licenses/gpl-3.0.html

#pragma endregion LICENSE

#pragma once

#ifndef PixelForge_BVH_CU
#define PixelForge_BVH_CU

#include <cuda_runtime>

#include "BVH.hpp"
#include "ray.hpp"

namespace pf
{
    __global__ void cuda_intersection(BVH& bvh, std::array<Ray3, 1 || 2 || 3 || 4> rays)
    {
        Node* d_node;
        Primitive* d_primitive;
        Ray3* d_rays;
        bool* d_results;
        int node_size = bvh.nodes.size();
        int ray_size = rays.size();

        hipMalloc(&d_nodes, node_size * sizeof(pf::Node));
        hipMalloc(&d_primitives, bvh.Primitives.size() * sizeof(pf::Primitive));
        hipMalloc(&d_rays, ray_size * sizeof(pf::Ray3));
        hipMalloc(&d_results, ray_size * sizeof(bool));
    
        hipMemcpy(d_nodes, bvh.nodes.data(), node_size * sizeof(pf::Node), hipMemcpyHostToDevice);
        hipMemcpy(d_primitives, bvh.primitives.data(), bvh.primitives.size() * sizeof(pf::Primitive), hipMemcpyHostToDevice);
        hipMemcpy(d_rays, rays.data(), ray_size * sizeof(pf::Ray), hipMemcpyHostToDevice);
    
        int threads_per_block = 256;
        int blocks_per_grid = (ray_size + threads_per_block - 1) / threads_per_block;
        intersectKernel<<<blocks_per_grid, threads_per_block>>>(d_nodes, d_primitives, d_rays, d_results, node_size, ray_size);
    
        std::array<bool, ray_size> results;
        hipMemcpy(results.data(), d_results, ray_size * sizeof(bool), hipMemcpyDeviceToHost);
    
        hipFree(d_nodes);
        hipFree(d_primitives);
        hipFree(d_rays);
        hipFree(d_results);
    
        for(int i = 0; i < results.size(); ++i)
        {
            if (results[i])
            {
                std::cout << "Ray " << i << " has intersected an object.\n";
            }
        }
    }
}

#endif // PixelForge_BVH_CU